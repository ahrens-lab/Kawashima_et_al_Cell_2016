#include "hip/hip_runtime.h"
// A C/CUDA code for registering XY translations of timeseries.
// Registration will be separately performed for individual Z-planes.

// This program assumes microscope output where binary 3D stacks are written sequentially 
//(T00000.stack,T00001.stack,...).  This extracts time series of a single z-plane from these
// 3D stacks, register time series to their average, and write out registered 
// time series into a new binary file.  This process is repeated for all the z-planes, and 
// separate binary files for each z-plane time series are created (Plane00.stack, Plane01.stack,...).
// 2D registration of individual time series is based on phase correlation algorithm.
// Calculation is accelerated by GPU computation implemented by CUDA library.
//
// For general instruction of building .cu file please refer to online resources.
// Work with Windows Server 2012, Visual Studio 2010, Tesla K20 board and CUDA toolkit v5.5.
// Please compile as x64 software.  Libtiff library is required (http://www.libtiff.org/)
//
// required data format
// Image Files		-> Series of 3D stack binary files (UINT16). T00000.stack,T00001.stack,...
// Image Dimension File -> "StackDimensions.bin". 
//                         This uint32 binary contains 3 values (width, height, number of z-planes).
//
// input 
// "the source directory"		-> The data directory. Space is not allowed as part of the name.
// "the max timepoint"			-> Maximum time point. For the attached sample enter 59.
// "the number of digits in file name"	-> If the file name is like T00001.stack, enter 5.
//
// output ("registered" folder in the data directory)
// ave.tif		-> Averaged 3D stack after registration.
// PlaneXX.stack	-> Registered time series of individual Z-planes.
// stackdim.txt		-> Dimensions of the stack (width, height,number of z-planes, time length).

// Developed by Takashi Kawashima, HHMI Janelia Research Campus
// Sept 19, 2016

#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <windows.h>
#include <process.h>
#include <io.h>
#include <ctime>
#include <string>
#include <sstream>
#include <iomanip>
#include <locale>
#include <direct.h>
#include <omp.h>
#include <math.h>
#include "tiffio.h"
#include "tiff.h"


using namespace std;

typedef struct ST{ 
	
	unsigned short* outstack;
	int* zlist;
	int* status;
	int zrepeat;
	int imglen;
	unsigned short **outpoints;
	string fullpatho;

}thestruct; 

string convert_int_to_fname(int framenum, string fname_head, string fname_tail,int numdigit);
string convert_int_to_oname(int framenum, string fname_head, string fname_tail,int numdigit,int tt);
char*   str2char(string inputname);
void write_output(void* s);
void write_tiff_file(string path, float* stack, int w, int h, int zlen);
string ReplaceString(std::string subject, const std::string& search,const std::string& replace);




__global__ void MakeComp1( hipfftDoubleComplex *a, hipfftDoubleReal *b, int imglen)
{
	int id = (int) gridDim.x*blockDim.x*blockIdx.y+blockIdx.x *  blockDim.x +  threadIdx.x;
	if(id <imglen){
	
	    a[id].x  =  b[id];
	    a[id].y  =  0;  
	}

}

__global__ void MakeComp2( hipfftDoubleComplex *a, unsigned short *b, unsigned short *c,int start,int imglen)
{
        int id = (int) gridDim.x*blockDim.x*blockIdx.y+blockIdx.x *  blockDim.x +  threadIdx.x;
        if(id <imglen){
            int id2= start+id;

            c[id]    =  b[id2];
            a[id].x  =  (double) b[id2];
            a[id].y  =  0;
        }
}


__global__ void ShiftPix( unsigned short *source, unsigned short *source2, int start, int shift,int imglen)
{
        int id = (int) gridDim.x*blockDim.x*blockIdx.y+blockIdx.x *  blockDim.x +  threadIdx.x ;
        if(id <imglen){    
            int id2 = (id+shift) % imglen;
            id2  += (id2 < 0 )*imglen;        
            source[start+id]  =  source2[id2];
        }
}
    

__global__ void KernelMult( hipfftDoubleComplex *a, hipfftDoubleComplex *b, hipfftDoubleComplex *c, int imglen)
{
        int id = (int) gridDim.x*blockDim.x*blockIdx.y+blockIdx.x *  blockDim.x +  threadIdx.x;
        if(id <imglen){
            c[id].x  =  a[id].x * b[id].x+a[id].y * b[id].y;
            c[id].y  =  a[id].x * b[id].y-a[id].y * b[id].x;
        }
}

int main()
{
		

	
	string str1="\\";
	string str2="\\\\";
	
	string sdir, tdir;
	int maxtime, reftime,digits;

	cout << "Please enter the source directory \n>>";
	cin  >> sdir;

	tdir=sdir;

	cout << "Please enter the max timepoint\n>>";
	cin  >> maxtime;

	cout << "Please specify the number of digits in file name\n>>";
	cin  >> digits;
	
	reftime=maxtime/2;

	string indir = ReplaceString(sdir,str1, str2);
	indir.append("\\\\");

	string outdir= ReplaceString(tdir,str1, str2);
	outdir.append("\\\\\\Registered\\\\");

	string fname_head = "T";
	string fname_tail = ".stack";
	string oname_head = "Plane";
	string oname_tail = ".stack";
	string avetif    = "ave.tif";
	string stackdim    = "stackdim.txt";
	
	__int64 max_memory=3200000000; // this depends on the GPU board
	

	int numthreads=2;
	time_t tstart, tend; 
	tstart = time(0);
	int passedtime;
	
	cout<<"Starting...\n";
	string fullpathi;		
	string fullpatho;  


	

	string dimname = "StackDimensions.bin";

	string dimpath = indir;
	dimpath.append(dimname);
	unsigned int tmp[3];

	FILE *fdim = fopen(str2char(dimpath),"rb");
		fread(tmp, sizeof(unsigned int),3,fdim);
	fclose(fdim);
	
	


	// copy log files////////////////////////////////////////////////////

	
	_mkdir(str2char(outdir));




	
	printf("width=%d, height=%d, z=%d\n",tmp[0],tmp[1],tmp[2]);	

	int width  = (int)tmp[0];
	int height = (int)tmp[1];
	int zmin=1;
	int zmax   = (int)tmp[2];
	
			
	int imglen= width*height;
	int stacklen= width*height*(zmax-zmin+1);
	__int64 singlelen = imglen*sizeof(unsigned short);	
	


	//define input and output array/////////////////////////////////////////////////////////////////////////////////
	
				

	unsigned short *stack;
	stack =  (unsigned short *)calloc((__int64)imglen*(__int64)(maxtime+1),sizeof(unsigned short));
	unsigned short *stack_ori =  stack;	
	
	unsigned short **stackpoints;
    	stackpoints=(unsigned short**)malloc((maxtime+1)*sizeof(unsigned short*));

	
	for (int ii=0;ii<maxtime+1;ii++){
		stackpoints[ii]=stack+(__int64)imglen*(__int64)ii;
	}
				
	unsigned short *outstack;
	outstack=  (unsigned short *)calloc((__int64)imglen*(__int64)(maxtime+1),sizeof(unsigned short));
	unsigned short *outstack_ori =  outstack;	

	
	//define reference array/////////////////////////////////////////////////////////////////////////


	unsigned short *ref_i;
	(unsigned short *)ref_i=  (unsigned short *) calloc((__int64)stacklen ,sizeof(unsigned short));
	unsigned short *ref_i_ori=ref_i;
	double *ref_d;
	(double*)ref_d =  (double *) calloc((__int64)stacklen, sizeof(double));
	double* ref_d_ori=ref_d;


	
	//define calculation array/////////////////////////////////////////////////////////////////////////////////

	float *avestack;
	avestack=  (float *)calloc((__int64)stacklen, sizeof(float));
	float *avestack_ori=avestack;

	double *ave_d;
	ave_d=  (double *)calloc((__int64)imglen, sizeof(double));
	double *ave_d_ori=ave_d;
	


	/// Setup CUDA //////////////////////////////////////////////////////////////////////////////////////////////////////
	

	hipSetDevice(1);
	hipDeviceReset();
    
	int  zstep= (int) (max_memory/((__int64)(imglen*sizeof(unsigned short))));
	int  zrepeat= (maxtime+1)/zstep;

	int  amari=(maxtime+1)-zstep*zrepeat;
	if (amari!=0){
		zrepeat +=1;
	}
        
	int* zlist;
	zlist=(int*)calloc(zrepeat,sizeof(int));
    
	for (int i=0;i<zrepeat; i++){
		if(amari != 0 && i == zrepeat-1){
			zlist[i]=amari;
		}
		else{
			zlist[i]=zstep;
		}
	}

	
	unsigned short **outpoints;
	outpoints = (unsigned short**)malloc(zrepeat * sizeof (unsigned short*));

	outpoints[0]=outstack_ori;
	if(zrepeat>1)
	{
		for(int ii=1;ii<zrepeat;ii++){
			outpoints[ii]=outpoints[ii-1]+zlist[ii-1]*imglen;
		}
	}
	
    
	unsigned short  *source00,*source0;
	hipfftDoubleReal *target0;
	hipfftDoubleComplex *target1, *source1, *target2, *source2, *mult1, *mult2;
	hipfftHandle fftPlan;
    
	hipMalloc((void**)&target0,  sizeof(hipfftDoubleReal)*imglen);
	hipMalloc((void**)&source0,  sizeof(unsigned short)*((__int64)imglen*(__int64)zlist[0]));
	hipMalloc((void**)&source00, sizeof(unsigned short)*imglen);

	hipMalloc((void**)&target1, sizeof(hipfftDoubleComplex)*imglen);
	hipMalloc((void**)&source1, sizeof(hipfftDoubleComplex)*imglen);       
	hipMalloc((void**)&target2, sizeof(hipfftDoubleComplex)*imglen);
	hipMalloc((void**)&source2, sizeof(hipfftDoubleComplex)*imglen);
	hipMalloc((void**)&mult1,   sizeof(hipfftDoubleComplex)*imglen);
	hipMalloc((void**)&mult2,   sizeof(hipfftDoubleComplex)*imglen);
    
    
    
	hipfftPlan2d(&fftPlan, height, width, HIPFFT_Z2Z);        
    
	hipblasHandle_t handle;     
	hipblasCreate(&handle);    
	hipError_t ErrorHandle;
		
    
	
	int threads_num = 64;
    
	int g2= (imglen /(64*threads_num))+1;
	dim3 grids(64,g2,1);   
    
    
	int row_shift, col_shift;
	int shift,start;
	int peakind ;
    
	
	//create reference  images///////////////////////////////////////////////////////////////////////////////////////////
	
	printf("Creating reference image for alignment...\n") ;
	double refc=0;
	int rs=0;
	size_t readlen;

	for(int r=-15;r<15; r++)
	{
		ref_d=ref_d_ori;
		ref_i=ref_i_ori;
		int rtime=reftime+r;
		rs=0;

		if (rtime>0 && rtime<maxtime);
		{
			refc +=1;
			string fname= convert_int_to_fname(rtime, fname_head, fname_tail, digits);
			fullpathi=indir;
			fullpathi.append(fname);
		
	
			FILE *fi=fopen(str2char(fullpathi),"rb");   
			if(fi==NULL)
			{
				cout << "Invalid FilePath";
				return 0;
			}

			while(rs==0){
				_lseeki64(_fileno(fi), 0, SEEK_SET);  
				readlen=fread(ref_i,sizeof(unsigned short),(__int64)stacklen,fi);
				if (readlen==(__int64)stacklen){rs=1;}
			}

			fclose(fi);

			for (int i=0;i<stacklen;i++){
				*ref_d += (double)*ref_i;
				ref_d++;ref_i++;
			}	
		}
	}


	ref_d=ref_d_ori;

	for (int i=0;i<stacklen;i++){
		*ref_d /=refc;
		ref_d++;
	}	

	ref_d=ref_d_ori;
	
	tend = time(0); 
	passedtime=(int)difftime(tend,tstart);
	printf("%d sec elapsed\n",passedtime) ;

	//Process images///////////////////////////////////////////////////////////////////////////////////////////

	int zplane;
	for (zplane=zmin-1;zplane< zmax; zplane++){

		string oname= convert_int_to_fname(zplane+1, oname_head, oname_tail, 2);
		fullpatho=outdir;
		fullpatho.append(oname);
		FILE *fo=fopen(str2char(fullpatho),"rb");   
		if(fo!=NULL)
		{
			fclose(fo);
			remove(str2char(fullpatho));
		}
		tend = time(0); 		
		passedtime=(int)difftime(tend,tstart);

	

		//Acquire reference image//////////////////////////////////////////////////////////////////////////////


		ref_d=ref_d_ori+(__int64)imglen*(__int64)zplane;

	
		//read stack image///////////////////////////////////////////////////////////////////////////
	
		
	    
		omp_set_num_threads(numthreads);

		#pragma omp parallel default(none) shared() firstprivate(imglen,zplane,fname_head,fname_tail,indir,stackpoints)
		{
			string fullpathi2;	
			string fname2;
			FILE *fi2 = NULL;
			size_t result;
		    int i;

			#pragma omp for 
			for (int tt=0;tt<maxtime+1;tt++){
				fname2= convert_int_to_fname(tt, fname_head, fname_tail, digits);
				fullpathi2=indir;
				fullpathi2.append(fname2);
				i=0;
				fi2=fopen(str2char(fullpathi2),"rb");   

				while(i==0){
					_lseeki64(_fileno(fi2), singlelen*(__int64) (zplane), SEEK_SET);  
					result=fread(stackpoints[tt],sizeof(unsigned short),imglen,fi2);
					if (result==imglen){i=1;}
				}

				fclose(fi2);	
			
			}
		}
		
		omp_set_num_threads(1);

		
		
		
		tend = time(0); 
		passedtime=(int)difftime(tend,tstart);
		printf("%d sec elapsed\n",passedtime) ;
		
			
		//compute withGPU/////////////////////////////////////////////////////////////////////////////////

		stack    =stack_ori;   
		outstack =outstack_ori;   
    
    
		hipMemcpy( target0,ref_d, sizeof(double)*imglen,hipMemcpyHostToDevice);
		MakeComp1 <<< grids,  threads_num >>> (target1, target0, imglen);
		hipfftExecZ2Z(fftPlan, target1, target2, HIPFFT_FORWARD);

	
		HANDLE myhandleA;
		
		int* status;
		status=(int *)calloc(1,sizeof(int));
		status[0]=-1;
		
		thestruct st;

		st.outstack = outstack;
		st.zlist    = zlist;
		st.status   = status;
		st.zrepeat  = zrepeat;
		st.imglen    = imglen;
		st.fullpatho   = fullpatho;
		st.outpoints = outpoints;
		

		myhandleA = (HANDLE)_beginthread(write_output, 0, (void *)&st);

	


        
		for (int ii=0;ii<zrepeat;ii++){
        
			start  = 0;

			hipMemcpyAsync(source0, stack, sizeof(unsigned short)*((__int64)imglen*(__int64)zlist[ii]),hipMemcpyHostToDevice);
			status[0]=status[0]+1;
				
			
			for (int zz=0;zz<zlist[ii]; zz++){

				MakeComp2 <<< grids,  threads_num >>> (source1, source0,source00,start,imglen);
				hipfftExecZ2Z(fftPlan, source1, source2, HIPFFT_FORWARD);
				KernelMult <<< grids,  threads_num >>> (target2, source2, mult1,imglen);

				hipfftExecZ2Z(fftPlan, mult1, mult2, HIPFFT_BACKWARD);
				hipblasIzamax(handle,imglen,mult2,1,&peakind);

				row_shift = (peakind-1) % width;
				col_shift = (peakind-1) / width ;

				if (row_shift > (width/2)){
					row_shift -=  width;
				}

				if (col_shift > (height/2)){
					col_shift -=  height;
				}
				shift = width*col_shift+row_shift;
				ShiftPix <<< grids, threads_num >>> (source0, source00,start, shift, imglen);
				start += imglen;

			}
			

  			hipMemcpyAsync(outpoints[ii], source0, sizeof(unsigned short)*imglen*zlist[ii],hipMemcpyDeviceToHost);
			stack          += imglen*zlist[ii];

		}

		ErrorHandle = hipGetLastError();
		status[0]=status[0]+1;

		
		
		
		tend = time(0); 
		passedtime=(int)difftime(tend,tstart);
		printf("%d sec elapsed\n",passedtime) ;
		
		WaitForSingleObject(myhandleA, INFINITE);


		// creating average image;///////////////////////////////////////////////////////

		outstack=outstack_ori;//outstack_ori;
		ave_d=ave_d_ori;

		for (int tt=0;tt<maxtime;tt++){

			for(int jj=0;jj<imglen;jj++){
				*ave_d += (double)*outstack;
				ave_d++;outstack++;
			}
			ave_d=ave_d_ori;
		}	
		outstack=outstack_ori;//outstack_o
		
		for(int jj=0;jj<imglen;jj++){
			*ave_d /= (double)maxtime;
			ave_d++;
		}
		ave_d=ave_d_ori;

		
		avestack=avestack_ori+imglen*zplane;

	
		for (int i=0;i<imglen;i++){
			*avestack = (float) *ave_d;
			avestack++; ave_d++;
		}	

			

		///disp time////////////////////////////////////////////////////////////////////
		

		tend = time(0); 
		passedtime=(int)difftime(tend,tstart);
		printf("Plane %d: Total %d sec elapsed\n",zplane+1,passedtime) ;
	}
		


	string avepath=outdir;
	avepath.append(avetif);

	avestack=avestack_ori;
	write_tiff_file(avepath, avestack, height, width, zmax);



	tend = time(0); 
	passedtime=(int)difftime(tend,tstart);
	printf("Total %d sec elapsed\n",passedtime) ;
	
	string dimmpath=outdir;
	dimmpath.append(stackdim);

	FILE *_fd=fopen(str2char(dimmpath),"w");	
	fprintf(_fd,"Y=%d \n",width);
	fprintf(_fd,"X=%d \n",height);
	fprintf(_fd,"Z=%d \n",maxtime);
	fclose(_fd);

	Sleep(5000);


	stack=stack_ori;
	outstack=outstack_ori;
	ref_i=ref_i_ori;
	ref_d=ref_d_ori;
	avestack=avestack_ori;
	ave_d=ave_d_ori;

	free(ref_i);
	free(ref_d);
	free(stack);
	free(outstack);
	free(avestack);
	free(ave_d);

    	hipfftDestroy(fftPlan);
    	hipFree(target0);
    	hipFree(target1);
    	hipFree(target2);
    	hipFree(source0);
	hipFree(source00);
    	hipFree(source1);
    	hipFree(source2);
    	hipFree(mult1);
    	hipFree(mult2);
    	hipblasDestroy(handle);  

    	return 0;
}

void write_output(void* s)
{	
	thestruct* t = (thestruct *)s;
	FILE *_fo=fopen(str2char(t -> fullpatho),"ab+");

	size_t result;
	size_t position=0;
	__int64 slen;  
	int ok=0;

	for (int z=0;z<( t -> zrepeat); z++)
	{
		while( t -> status[0] <= z){
			Sleep(10);
		}
		slen=(__int64)(t -> imglen)*(__int64)(t -> zlist[z]);
		result=fwrite(t -> outpoints[z], sizeof(unsigned short),slen,_fo);

		if(result != slen)
		{

			while(ok==0)
			{
				_lseeki64(_fileno(_fo), position, SEEK_SET); 
				result=fwrite(t -> outpoints[z], sizeof(unsigned short),slen,_fo); 
				if(result ==slen){ok=1;};
			}
			
		}

		position=position+slen;
		ok=0;
	}

	fclose(_fo);
	_endthread();
}

string  convert_int_to_fname(int framenum, string fname_head, string fname_tail,int numdigit){


	
	ostringstream Convert;

	string Result, out;
	Convert << setw(numdigit) << setfill('0') << framenum;
	Result=Convert.str();

	out=fname_head;
	out.append(Result);
	out.append(fname_tail);

	return out;
}

string  convert_int_to_oname(int framenum, string fname_head, string fname_tail,int numdigit, int num){
	
	ostringstream Convert;

	string Result,out;
	Convert << setw(numdigit) << setfill('0') << framenum;
	Convert << setw(2) << setfill('0') << num;

	Result=Convert.str();

	out=fname_head;
	out.append(Result);
	out.append(fname_tail);

	return out;
}


char* str2char(string input){

	string search="\\";
	string replace="\\\\";
	size_t pos = 0;
	while ((pos = input.find(search, pos)) != std::string::npos) {
			input.replace(pos, search.length(), replace);
			pos += replace.length();
	}

	char *out=new char[input.length() +1];
	strcpy(out,input.c_str());

	return out;
}



string ReplaceString(std::string subject, const std::string& search,const std::string& replace) {
	size_t pos = 0;
	while ((pos = subject.find(search, pos)) != std::string::npos) {
			subject.replace(pos, search.length(), replace);
			pos += replace.length();
	}
	return subject;


}

void write_tiff_file(string path, float* stack, int w, int h, int zlen){
	
	TIFF *tif;

    unsigned short *pdst, *pdst_ori;
    pdst=(unsigned short*) calloc((size_t)w, sizeof (unsigned short));
	pdst_ori=pdst;

	
	tif=TIFFOpen(str2char(path),"wb");
	for (int pp=0;pp<zlen;pp++)
    {
        TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, w);
        TIFFSetField(tif, TIFFTAG_IMAGELENGTH, h);
        TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, 16);
        TIFFSetField(tif, TIFFTAG_COMPRESSION, 1);
        TIFFSetField(tif, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_MINISBLACK);
        TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 1);
        TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, 1);
        TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
        TIFFSetField(tif, TIFFTAG_RESOLUTIONUNIT, RESUNIT_NONE);
        TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);
        TIFFSetField(tif, TIFFTAG_SUBFILETYPE, FILETYPE_PAGE);
        TIFFSetField(tif, TIFFTAG_PAGENUMBER, (unsigned short) pp, (unsigned short) zlen);

        __int64 zstart    = (__int64)pp*w*h;
		int cstart;
        // now go line by line to write out the image data

        pdst=pdst_ori;
        for (int row = 0; row < h; row++ )
        {
			cstart=row+zstart;

            for (int col = 0; col < w; col++){
                *pdst++ = (unsigned short) stack[cstart+h*col];
            }
            pdst=pdst_ori;
            TIFFWriteScanline(tif, pdst, row, 0);
        }

        TIFFWriteDirectory(tif);
    }

    TIFFClose(tif);
}













